#include "hip/hip_runtime.h"
#include "foo.h"
#include <stdio.h>

__global__ void hello() {
    printf("Test from GPU\n");
}

void useHello() {
    hello<<<1, 5>>>();
    hipDeviceSynchronize();
    //hipDeviceReset();
}